#include "hip/hip_runtime.h"
/***************************************************************************
** intoPIX SA & Fraunhofer IIS (hereinafter the "Software Copyright       **
** Holder") hold or have the right to license copyright with respect to   **
** the accompanying software (hereinafter the "Software").                **
**                                                                        **
** Copyright License for Evaluation and Testing                           **
** --------------------------------------------                           **
**                                                                        **
** The Software Copyright Holder hereby grants, to any implementer of     **
** this ISO Standard, an irrevocable, non-exclusive, worldwide,           **
** royalty-free, sub-licensable copyright licence to prepare derivative   **
** works of (including translations, adaptations, alterations), the       **
** Software and reproduce, display, distribute and execute the Software   **
** and derivative works thereof, for the following limited purposes: (i)  **
** to evaluate the Software and any derivative works thereof for          **
** inclusion in its implementation of this ISO Standard, and (ii)         **
** to determine whether its implementation conforms with this ISO         **
** Standard.                                                              **
**                                                                        **
** The Software Copyright Holder represents and warrants that, to the     **
** best of its knowledge, it has the necessary copyright rights to        **
** license the Software pursuant to the terms and conditions set forth in **
** this option.                                                           **
**                                                                        **
** No patent licence is granted, nor is a patent licensing commitment     **
** made, by implication, estoppel or otherwise.                           **
**                                                                        **
** Disclaimer: Other than as expressly provided herein, (1) the Software  **
** is provided �AS IS� WITH NO WARRANTIES, EXPRESS OR IMPLIED, INCLUDING  **
** BUT NOT LIMITED TO, THE WARRANTIES OF MERCHANTABILITY, FITNESS FOR A   **
** PARTICULAR PURPOSE AND NON-INFRINGMENT OF INTELLECTUAL PROPERTY RIGHTS **
** and (2) neither the Software Copyright Holder (or its affiliates) nor  **
** the ISO shall be held liable in any event for any damages whatsoever   **
** (including, without limitation, damages for loss of profits, business  **
** interruption, loss of information, or any other pecuniary loss)        **
** arising out of or related to the use of or inability to use the        **
** Software.�                                                             **
**                                                                        **
** RAND Copyright Licensing Commitment                                    **
** -----------------------------------                                    **
**                                                                        **
** IN THE EVENT YOU WISH TO INCLUDE THE SOFTWARE IN A CONFORMING          **
** IMPLEMENTATION OF THIS ISO STANDARD, PLEASE BE FURTHER ADVISED THAT:   **
**                                                                        **
** The Software Copyright Holder agrees to grant a copyright              **
** license on reasonable and non- discriminatory terms and conditions for **
** the purpose of including the Software in a conforming implementation   **
** of the ISO Standard. Negotiations with regard to the license are       **
** left to the parties concerned and are performed outside the ISO.       **
**                                                                        **
** No patent licence is granted, nor is a patent licensing commitment     **
** made, by implication, estoppel or otherwise.                           **
***************************************************************************/
#include "libjxs.h"
#include "common.h"
#include <assert.h>
#include <hip/hip_runtime.h>
#include "nlt.cuh"
__device__ void clamp(xs_data_in_t v, xs_data_in_t max_v, xs_data_in_t *ret)
{
    if (v > max_v)
    {
        v = max_v;
    }
    if (v < 0)
    {
        v = 0;
    }
    *ret = v;
}

__device__ void clamp64(int64_t v, int64_t max_v, int64_t *ret)
{
    if (v > max_v)
    {
        v = max_v;
    }
    if (v < 0)
    {
        v = 0;
    }
    *ret = v;
}

__global__ void kernel_nlt_inverse_linear(xs_image_t *gpu_image,
                                          const uint8_t s, const xs_data_in_t dclev_and_rounding,
                                          const xs_data_in_t max_val)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int pixels_num = 0;
    for (int c = 0; c < gpu_image->ncomps; ++c)
    {
        pixels_num = (size_t)(gpu_image->width / gpu_image->sx[c]) * (size_t)(gpu_image->height / gpu_image->sy[c]);
        if (tid < pixels_num)
        {
            xs_data_in_t *the_ptr = gpu_image->comps_array[c] + tid;
            clamp((*the_ptr + dclev_and_rounding) >> s, max_val, the_ptr);
            break;
        }
        tid -= pixels_num;
    }
}

void gpu_nlt_inverse_linear(xs_image_t *image, xs_image_t *gpu_image, const uint8_t Bw)
{
    const uint8_t s = Bw - (uint8_t)image->depth;
    const xs_data_in_t dclev_and_rounding = ((1 << Bw) >> 1) + ((1 << s) >> 1);
    const xs_data_in_t max_val = (1 << image->depth) - 1;
    int pixels_num = 0;
    for (int c = 0; c < image->ncomps; ++c)
    {
        pixels_num += (size_t)(image->width / image->sx[c]) * (size_t)(image->height / image->sy[c]);
    }
    const int block_size = BLOCK_SIZE;
    const int grid_size = (pixels_num + block_size - 1) / block_size;
    kernel_nlt_inverse_linear<<<grid_size, block_size>>>(gpu_image, s, dclev_and_rounding, max_val);
}
__global__ void kernel_nlt_inverse_quadratic(xs_image_t *gpu_image, const xs_data_in_t dclev,
                                             const uint8_t s, const xs_data_in_t vdco, const xs_data_in_t s_r,
                                             const xs_data_in_t max_val, const xs_data_in_t max_coef)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int pixels_num = 0;
    for (int c = 0; c < gpu_image->ncomps; ++c)
    {
        pixels_num = (size_t)(gpu_image->width / gpu_image->sx[c]) * (size_t)(gpu_image->height / gpu_image->sy[c]);
        if (tid < pixels_num)
        {
            xs_data_in_t *the_ptr = gpu_image->comps_array[c] + tid;
            int64_t v;
            clamp(*the_ptr + dclev, max_coef, (xs_data_in_t *)&v);
            clamp64(((v * v + s_r) >> s) + vdco, max_val, (int64_t *)the_ptr);
            break;
        }
        tid -= pixels_num;
    }
}

void gpu_nlt_inverse_quadratic(xs_image_t *image, xs_image_t *gpu_image, const uint8_t Bw, const xs_nlt_parameters_t nlt_parameters)
{
    const xs_data_in_t vdco = (xs_data_in_t)nlt_parameters.quadratic.alpha - (xs_data_in_t)nlt_parameters.quadratic.sigma * 32768;
    const uint8_t s = (Bw << 1) - (uint8_t)image->depth;
    const xs_data_in_t dclev = ((1 << Bw) >> 1);
    const xs_data_in_t s_r = (1 << s) >> 1;
    const xs_data_in_t max_val = (1 << image->depth) - 1;
    const xs_data_in_t max_coef = (1 << Bw) - 1;
    int pixels_num = 0;
    for (int c = 0; c < image->ncomps; ++c)
    {
        pixels_num += (size_t)(image->width / image->sx[c]) * (size_t)(image->height / image->sy[c]);
    }
    const int block_size = BLOCK_SIZE;
    const int grid_size = (pixels_num + block_size - 1) / block_size;
    kernel_nlt_inverse_quadratic<<<grid_size, block_size>>>(gpu_image, dclev, s, vdco, s_r, max_val, max_coef);
}

__global__ void kernel_nlt_inverse_extended(xs_image_t *gpu_image, const uint8_t Bw, const uint8_t e,
                                            uint32_t T1, uint32_t T2, const uint8_t s,
                                            const xs_data_in_t s_r, const xs_data_in_t max_val,
                                            const xs_data_in_t dclev, const xs_data_in_t max_coef,
                                            const int64_t A1, const int64_t A3,
                                            const int64_t B1, const int64_t B2, const int64_t B3)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int pixels_num = 0;
    for (int c = 0; c < gpu_image->ncomps; ++c)
    {
        pixels_num = (size_t)(gpu_image->width / gpu_image->sx[c]) * (size_t)(gpu_image->height / gpu_image->sy[c]);
        if (tid < pixels_num)
        {
            xs_data_in_t *the_ptr = gpu_image->comps_array[c] + tid;
            int64_t v = (int64_t)*the_ptr + dclev;
            if (v < T1)
            {
                clamp64(B1 - v, max_coef, &v);
                v = A1 - v * v;
            }
            else if (v < T2)
            {
                v = (v << e) + B2;
            }
            else
            {
                clamp64(v - B3, max_coef, &v);
                v = A3 + v * v;
            }
            clamp64((v + s_r) >> s, max_val, (int64_t *)the_ptr);
            break;
        }
        tid -= pixels_num;
    }
}

void gpu_nlt_inverse_extended(xs_image_t *image, xs_image_t *gpu_image, const uint8_t Bw, const xs_nlt_parameters_t nlt_parameters)
{
    const uint8_t e = Bw - nlt_parameters.extended.E;
    const int64_t B2 = (int64_t)nlt_parameters.extended.T1 * nlt_parameters.extended.T1;
    const int64_t A1 = B2 + ((int64_t)nlt_parameters.extended.T1 << e) + (1ll << (2ll * e - 2));
    const int64_t B1 = (int64_t)nlt_parameters.extended.T1 + (1ll << (e - 1));
    const int64_t A3 = B2 + ((int64_t)nlt_parameters.extended.T2 << e) - (1ll << (2ll * e - 2));
    const int64_t B3 = (int64_t)nlt_parameters.extended.T2 - (1ll << (e - 1));
    const uint8_t s = (Bw << 1) - (uint8_t)image->depth;
    const xs_data_in_t s_r = (1 << s) >> 1;
    const xs_data_in_t max_val = (1 << image->depth) - 1;
    const xs_data_in_t max_coef = (1 << Bw) - 1;
    const xs_data_in_t dclev = ((1 << Bw) >> 1);
    int pixels_num = 0;
    for (int c = 0; c < image->ncomps; ++c)
    {
        pixels_num += (size_t)(image->width / image->sx[c]) * (size_t)(image->height / image->sy[c]);
    }
    const int block_size = BLOCK_SIZE;
    const int grid_size = (pixels_num + block_size - 1) / block_size;
    kernel_nlt_inverse_extended<<<grid_size, block_size>>>(gpu_image, Bw, e, nlt_parameters.extended.T1, nlt_parameters.extended.T2, s, s_r, max_val, dclev, max_coef, A1, A3, B1, B2, B3);
}

void gpu_nlt_inverse_transform(xs_image_t *image, xs_image_t *gpu_image, const xs_config_parameters_t *p)
{
    switch (p->Tnlt)
    {
    case XS_NLT_NONE:
    {
        gpu_nlt_inverse_linear(image, gpu_image, p->Bw);
        break;
    }
    case XS_NLT_QUADRATIC:
    {
        gpu_nlt_inverse_quadratic(image, gpu_image, p->Bw, p->Tnlt_params);
        break;
    }
    case XS_NLT_EXTENDED:
    {
        gpu_nlt_inverse_extended(image, gpu_image, p->Bw, p->Tnlt_params);
        break;
    }
    default:
        assert(false);
        break;
    }
}
