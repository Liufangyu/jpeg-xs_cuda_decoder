#include "hip/hip_runtime.h"
#include "mct.cuh"

#include "common.h"
#include <assert.h>
#include <malloc.h>
#include <stdio.h>

__device__ __host__ void swap_ptr(xs_data_in_t **p1, xs_data_in_t **p2)
{
    xs_data_in_t *tmp = *p1;
    *p1 = *p2;
    *p2 = tmp;
}

__global__ void kernel_mct_inverse_rct(xs_image_t *gpu_image, uint32_t len)
{

    const uint32_t tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < len)
    {
        xs_data_in_t *c0 = gpu_image->comps_array[0] + tid;
        xs_data_in_t *c1 = gpu_image->comps_array[1] + tid;
        xs_data_in_t *c2 = gpu_image->comps_array[2] + tid;
        const xs_data_in_t tmp = *c0 - ((*c1 + *c2) >> 2);
        *c0 = tmp + *c2;
        *c2 = tmp + *c1;
        *c1 = tmp;
        ++c0;
        ++c1;
        ++c2;
    }
}
void gpu_mct_inverse_rct(xs_image_t *image, xs_image_t *gpu_image)
{
    const uint32_t len = image->width * image->height;

    const int block_size = BLOCK_SIZE;
    const int grid_size = (len + block_size - 1) / block_size;
    kernel_mct_inverse_rct<<<grid_size, block_size>>>(gpu_image, len);
}

__device__ __host__ void mct_tetrix_access(xs_image_t *im, const int c, const int Cf, const int Ct, const int rx, const int ry, const int x, const int y, xs_data_in_t *ret)
{
    // Stupid magic.
    assert(c >= 0 && c <= 3);
    assert(Cf == 0 || Cf == 3);
    assert(Ct == 0 || Ct == 1);
    assert(rx >= -1 && rx <= 1);
    assert(ry >= -1 && ry <= 1);
    int t_x = rx + ((Ct + c) & 1);
    int t_y = ry + (((~(c)) >> 1) & 1);
    assert(t_x >= -1 && t_x <= 2);
    assert(t_y >= -1 && t_y <= 2);
    const int k = ((((~(t_y)) << 1) & 2) | (((Ct) ^ (t_x)) & 1));
    assert(k >= 0 && k <= 3);
    if (Cf == 3)
    {
        t_y &= 1;
    }
    t_x += x << 1;
    t_y += y << 1;
    if (t_x < 0)
    {
        t_x += 2;
    }
    else if (t_x >= (im->width << 1))
    {
        t_x -= 2;
    }
    if (t_y < 0)
    {
        t_y += 2;
    }
    else if (t_y >= (im->height << 1))
    {
        t_y -= 2;
    }
    t_x >>= 1;
    t_y >>= 1;
    assert(t_x >= 0 && t_x < im->width);
    assert(t_y >= 0 && t_y < im->height);
    // assert(mct_tetrix_access_slow(im, c, Cf, Ct, rx, ry, x, y) == (im->comps_array[k] + t_y * im->width + t_x));
    *ret = im->comps_array[k][(size_t)t_y * (size_t)im->width + t_x];
}
__global__ void kernel_inverse_average(xs_image_t *gpu_image, int Cf, int Ct, int width, uint32_t len)
{
    const uint32_t tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < len)
    {
        const int x = tid % width;
        const int y = tid / width;
        xs_data_in_t dtl, dtr, dbl, dbr;
        mct_tetrix_access(gpu_image, 0, Cf, Ct, -1, -1, x, y, &dtl);
        mct_tetrix_access(gpu_image, 0, Cf, Ct, 1, -1, x, y, &dtr);
        mct_tetrix_access(gpu_image, 0, Cf, Ct, -1, 1, x, y, &dbl);
        mct_tetrix_access(gpu_image, 0, Cf, Ct, 1, 1, x, y, &dbr);
        gpu_image->comps_array[0][tid] -= (dtl + dtr + dbl + dbr) >> 3;
    }
}
__global__ void kernel_inverse_delta(xs_image_t *gpu_image, int Cf, int Ct, int width, uint32_t len)
{
    const uint32_t tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < len)
    {
        const int x = tid % width;
        const int y = tid / width;
        xs_data_in_t ytl, ytr, ybl, ybr;
        mct_tetrix_access(gpu_image, 3, Cf, Ct, -1, -1, x, y, &ytl);
        mct_tetrix_access(gpu_image, 3, Cf, Ct, 1, -1, x, y, &ytr);
        mct_tetrix_access(gpu_image, 3, Cf, Ct, -1, 1, x, y, &ybl);
        mct_tetrix_access(gpu_image, 3, Cf, Ct, 1, 1, x, y, &ybr);
        gpu_image->comps_array[3][tid] += (ytl + ytr + ybl + ybr) >> 2;
    }
}

__global__ void kernel_inverse_Y(xs_image_t *gpu_image, int Cf, int Ct, int e1, int e2, int width, uint32_t len)
{
    const uint32_t tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < len)
    {
        const int x = tid % width;
        const int y = tid / width;

        xs_data_in_t bl, br, rt, rb;
        mct_tetrix_access(gpu_image, 0, Cf, Ct, -1, 0, x, y, &bl);
        mct_tetrix_access(gpu_image, 0, Cf, Ct, 1, 0, x, y, &br);
        mct_tetrix_access(gpu_image, 0, Cf, Ct, 0, -1, x, y, &rt);
        mct_tetrix_access(gpu_image, 0, Cf, Ct, 0, 1, x, y, &rb);
        gpu_image->comps_array[0][tid] -= (((bl + br) << e2) + ((rt + rb) << e1)) >> 3;

        xs_data_in_t bt, bb, rl, rr;

        mct_tetrix_access(gpu_image, 3, Cf, Ct, 0, -1, x, y, &bt);
        mct_tetrix_access(gpu_image, 3, Cf, Ct, 0, 1, x, y, &bb);
        mct_tetrix_access(gpu_image, 3, Cf, Ct, -1, 0, x, y, &rl);
        mct_tetrix_access(gpu_image, 3, Cf, Ct, 1, 0, x, y, &rr);
        gpu_image->comps_array[3][tid] -= (((bt + bb) << e2) + ((rl + rr) << e1)) >> 3;
    }
}

__global__ void kernel_inverse_CbCr(xs_image_t *gpu_image, int Cf, int Ct, int e1, int e2, int width, uint32_t len)
{
    const uint32_t tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < len)
    {
        const int x = tid % width;
        const int y = tid / width;

        xs_data_in_t gl, gr, gt, gb;
        mct_tetrix_access(gpu_image, 1, Cf, Ct, -1, 0, x, y, &gl);
        mct_tetrix_access(gpu_image, 1, Cf, Ct, 1, 0, x, y, &gr);
        mct_tetrix_access(gpu_image, 1, Cf, Ct, 0, -1, x, y, &gt);
        mct_tetrix_access(gpu_image, 1, Cf, Ct, 0, 1, x, y, &gb);

        gpu_image->comps_array[1][tid] += (gl + gr + gt + gb) >> 2;

        mct_tetrix_access(gpu_image, 2, Cf, Ct, 0, -1, x, y, &gl);
        mct_tetrix_access(gpu_image, 2, Cf, Ct, 0, 1, x, y, &gr);
        mct_tetrix_access(gpu_image, 2, Cf, Ct, -1, 0, x, y, &gt);
        mct_tetrix_access(gpu_image, 2, Cf, Ct, 1, 0, x, y, &gb);
        gpu_image->comps_array[2][tid] += (gl + gr + gt + gb) >> 2;
    }
}

__global__ void gpu_swap_image_ptr(xs_image_t *gpu_image)
{
    const int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid == 0)
    {
        swap_ptr(&gpu_image->comps_array[0], &gpu_image->comps_array[2]);
        swap_ptr(&gpu_image->comps_array[1], &gpu_image->comps_array[3]);
    }
}
void gpu_mct_inverse_tetrix(xs_image_t *im, xs_image_t *gpu_image, const xs_cfa_pattern_t cfa_pattern, const xs_cts_parameters_t cts_parameters)
{
    assert(im->ncomps == 4);
    assert(im->sx[0] == im->sx[1] && im->sx[0] == im->sx[2] && im->sx[0] == im->sx[3]);
    assert(im->sy[0] == im->sy[1] && im->sy[0] == im->sy[2] && im->sy[0] == im->sy[3]);

    const int Cf = cts_parameters.Cf;
    const int Ct = (cfa_pattern == XS_CFA_RGGB || cfa_pattern == XS_CFA_BGGR) ? 0 : 1;
    const uint8_t e1 = cts_parameters.e1;
    const uint8_t e2 = cts_parameters.e2;

    // Inverse average.
    const int len = im->height * im->width;
    const int block_size = BLOCK_SIZE;
    const int grid_size = (len + block_size - 1) / block_size;
    kernel_inverse_average<<<grid_size, block_size>>>(gpu_image, Cf, Ct, im->width, len);
    hipDeviceSynchronize();
    // Inverse delta.

    kernel_inverse_delta<<<grid_size, block_size>>>(gpu_image, Cf, Ct, im->width, len);
    hipDeviceSynchronize();
    // Inverse Y.

    kernel_inverse_Y<<<grid_size, block_size>>>(gpu_image, Cf, Ct, e1, e2, im->width, len);
    hipDeviceSynchronize();
    // Inverse CbCr.
    kernel_inverse_CbCr<<<grid_size, block_size>>>(gpu_image, Cf, Ct, e1, e2, im->width, len);
    hipDeviceSynchronize();
    // Reassign component order.
    swap_ptr(&im->comps_array[0], &im->comps_array[2]);
    swap_ptr(&im->comps_array[1], &im->comps_array[3]);
    gpu_swap_image_ptr<<<1, 1>>>(gpu_image);
    hipDeviceSynchronize();
}

void gpu_mct_inverse_transform(xs_image_t *image, xs_image_t *gpu_image, const xs_config_parameters_t *p)
{
    switch (p->color_transform)
    {
    case XS_CPIH_NONE:
    {
        break;
    }
    case XS_CPIH_RCT:
    {
        gpu_mct_inverse_rct(image, gpu_image);
        break;
    }
    case XS_CPIH_TETRIX:
    {
        gpu_mct_inverse_tetrix(image, gpu_image, p->cfa_pattern, p->tetrix_params);
        break;
    }
    default:
        assert(!"Unknown color transform");
        break;
    }
}
