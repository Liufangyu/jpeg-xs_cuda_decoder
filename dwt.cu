#include "hip/hip_runtime.h"
#include "dwt.cuh"
#include <assert.h>
#include <hip/hip_runtime.h>
#include "libjxs.h"
#include "ids.h"
typedef enum dwt_mode_t
{
    VERTICAL,
    HORIZONTAL,
} dwt_mode_t;

__global__ void kernel_dwt_inverse_low_pass(xs_data_in_t *const base, xs_data_in_t *const end, const ptrdiff_t x_inc, const ptrdiff_t y_inc,
                                            const int column_num, const dwt_mode_t mode,
                                            const int bound, const uint32_t len)
{

    const uint32_t tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < len)
    {
        const int column = tid % column_num;
        const int raw = tid / column_num;
        xs_data_in_t *p = base;
        if (mode == VERTICAL)
        {

            if (raw == 0)
            {
                p += x_inc * column;
                if (p < end)
                {
                    *p -= (*(p + y_inc) + 1) >> 1;
                }
            }
            else
            {
                p += (y_inc << 1) * raw;
                if (p - base < bound - y_inc)
                {
                    p += x_inc * column;
                    if (p < end)
                    {
                        *p -= (*(p - y_inc) + *(p + y_inc) + 2) >> 2;
                    }
                }
                else if (p - base < bound)
                {
                    p += x_inc * column;
                    if (p < end)
                    {
                        *p -= (*(p - y_inc) + 1) >> 1;
                    }
                }
            }
        }
        else if (mode == HORIZONTAL)
        {

            if (column == 0)
            {
                p += y_inc * raw;
                if (p < end)
                {
                    *p -= (*(p + x_inc) + 1) >> 1;
                }
            }
            else
            {
                p += (x_inc << 1) * column;
                if (p - base < bound - x_inc)
                {
                    p += y_inc * raw;
                    if (p < end)
                    {
                        *p -= (*(p - x_inc) + *(p + x_inc) + 2) >> 2;
                    }
                }
                else if (p - base < bound)
                {
                    p += y_inc * raw;
                    if (p < end)
                    {
                        *p -= (*(p - x_inc) + 1) >> 1;
                    }
                }
            }
        }
    }
}

/*


 dwt的边界还没有计算
*/
__global__ void kernel_dwt_inverse_high_pass(xs_data_in_t *const base, xs_data_in_t *const end, const ptrdiff_t x_inc, const ptrdiff_t y_inc,
                                             const int column_num, const dwt_mode_t mode,
                                             const int bound, const uint32_t len)
{

    const uint32_t tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < len)
    {
        const int column = tid % column_num;
        const int raw = tid / column_num;

        if (mode == VERTICAL)
        {
            xs_data_in_t *p = base + y_inc;
            p += (y_inc << 1) * raw;
            if (p - base < bound - y_inc)
            {
                p += x_inc * column;
                if (p < end)
                {
                    *p += (*(p - y_inc) + *(p + y_inc)) >> 1;
                }
            }
            else if (p - base < bound)
            {
                p += x_inc * column;
                if (p < end)
                {
                    *p += *(p - y_inc);
                }
            }
        }
        else if (mode == HORIZONTAL)
        {
            xs_data_in_t *p = base + x_inc;
            p += (x_inc << 1) * column;
            if (p - base < bound - x_inc)
            {
                p += y_inc * raw;
                if (p < end)
                {
                    *p += (*(p - x_inc) + *(p + x_inc)) >> 1;
                }
            }
            else if (p - base < bound)
            {
                p += y_inc * raw;
                if (p < end)
                {
                    *p += *(p - x_inc);
                }
            }
        }
    }
}

void gpu_dwt_inverse_horizontal(const ids_t *ids, xs_data_in_t **gpu_comps_array, const int k, const int h_level, const int v_level, hipStream_t *stream)
{

    const ptrdiff_t x_inc = (ptrdiff_t)1 << h_level;
    const ptrdiff_t y_inc = (ptrdiff_t)ids->comp_w[k] << v_level;
    xs_data_in_t *base = gpu_comps_array[k];
    xs_data_in_t *const end = base + (size_t)ids->comp_w[k] * (size_t)ids->comp_h[k];
    const int column_num = ((ids->comp_w[k] + x_inc - 1) / x_inc);
    const int raw_num = (((size_t)ids->comp_w[k] * (size_t)ids->comp_h[k] + y_inc - 1) / y_inc);
    const int threads_num = raw_num * column_num;
    int block_size = BLOCK_SIZE;
    const int grid_size = (threads_num + block_size - 1) / block_size;
    kernel_dwt_inverse_low_pass<<<grid_size, block_size, 0, stream[k]>>>(base, end, x_inc, y_inc, column_num, HORIZONTAL, ids->comp_w[k], threads_num);
    kernel_dwt_inverse_high_pass<<<grid_size, block_size, 0, stream[k]>>>(base, end, x_inc, y_inc, column_num, HORIZONTAL, ids->comp_w[k], threads_num);
}
void gpu_dwt_inverse_vertical_(const ids_t *ids, xs_data_in_t **gpu_comps_array, const int k, const int h_level, const int v_level, hipStream_t *stream)
{

    const ptrdiff_t x_inc = (ptrdiff_t)1 << h_level;
    const ptrdiff_t y_inc = (ptrdiff_t)ids->comp_w[k] << v_level;
    xs_data_in_t *base = gpu_comps_array[k];
    xs_data_in_t *const end = base + (size_t)ids->comp_w[k] * (size_t)ids->comp_h[k];
    const int column_num = ((ids->comp_w[k] + x_inc - 1) / x_inc);
    const int raw_num = (((size_t)ids->comp_w[k] * (size_t)ids->comp_h[k] + y_inc - 1) / y_inc);
    const int threads_num = raw_num * column_num;
    int block_size = BLOCK_SIZE;
    const int grid_size = (threads_num + block_size - 1) / block_size;
    kernel_dwt_inverse_low_pass<<<grid_size, block_size, 0, stream[k]>>>(base, end, x_inc, y_inc, column_num, VERTICAL, (size_t)ids->comp_w[k] * (size_t)ids->comp_h[k], threads_num);
    kernel_dwt_inverse_high_pass<<<grid_size, block_size, 0, stream[k]>>>(base, end, x_inc, y_inc, column_num, VERTICAL, (size_t)ids->comp_w[k] * (size_t)ids->comp_h[k], threads_num);
}

void gpu_dwt_inverse_transform(const ids_t *ids, xs_data_in_t **gpu_comps_array)
{
    hipStream_t streams[N_STREAM];
    for (int i = 0; i < N_STREAM; i++)
    {
        hipStreamCreate(streams + i);
    }
    for (int k = 0; k < ids->ncomps - ids->sd; ++k)
    {
        assert(ids->nlxyp[k].y <= ids->nlxyp[k].x);

        for (int d = ids->nlxyp[k].x - 1; d >= ids->nlxyp[k].y; --d)
        {
            gpu_dwt_inverse_horizontal(ids, gpu_comps_array, k, d, ids->nlxyp[k].y, streams);
        }

        for (int d = ids->nlxyp[k].y - 1; d >= 0; --d)
        {
            gpu_dwt_inverse_horizontal(ids, gpu_comps_array, k, d, d, streams);
            gpu_dwt_inverse_vertical_(ids, gpu_comps_array, k, d, d, streams);
        }
    }
    for (int i = 0; i < N_STREAM; i++)
    {
        hipStreamDestroy(streams[i]);
    }
}
