#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "buf_mgmt.h"
#include "gcli_budget.h"
#include "pred.h"
#include "bitpacking.h"
#include "precinct.h"
#include "sigbuffer.h"
#include "gcli_methods.h"
#include "rate_control.h"
#include "packing.cuh"
#include <malloc.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <stdint.h>
#include <assert.h>

#include "libjxs.h"
#include "xs_config.h"
#include "xs_markers.h"
#include "common.h"
#include "precinct.h"
#include "buf_mgmt.h"
#include "bitpacking.h"
#include "budget.h"
#include "packing.h"
#include "quant.h"
#include "ids.h"
#include "dwt.h"
#include "mct.h"
#include "nlt.h"

/*

nvcc bitpacking.c budget.c buf_mgmt.c data_budget.c dwt.c gcli_budget.c gcli_methods.c ids.c image.c mct.c nlt.c packing.c precinct.c precinct_budget.c precinct_budget_table.c pred.c predbuffer.c quant.c rate_control.c sb_weighting.c sig_flags.c sigbuffer.c version.c xs_config.c xs_config_parser.c xs_dec.c xs_markers.c packing.cu xs_dec_main.c file_io.c cmdline_options.c file_sequence.c image_open.c v210.c rgb16.c yuv16.c planar.c uyvy8.c argb.c mono.c ppm.c pgx.c helpers.c -o jpegxs_decoder -w -rdc=true -gencode=arch=compute_61,code=compute_61
*/

#define SIGFLAGS_NEXTLVL_SIZE(w, g) (((w) + (g)-1) / (g))
#define MAXB (sizeof(uint64_t) * 8)

__global__ void kernel_convert_ipx_htobe64(uint64_t *bitstream_ptr, uint64_t len)
{
    const uint64_t tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < len)
    {
        uint64_t in = bitstream_ptr[tid];
        union
        {
            uint64_t integer;
            uint8_t bytes[8];
        } a, b;
        a.integer = in;
        for (int i = 0; i < 8; i++)
            b.bytes[i] = a.bytes[7 - i];
        bitstream_ptr[tid] = b.integer;
    }
}

void gpu_convert_ipx_htobe64(uint64_t *bitstream_ptr, uint64_t max_size)
{
    const int block_size = BLOCK_SIZE;
    const int grid_size = (max_size + block_size - 1) / block_size;
    kernel_convert_ipx_htobe64<<<grid_size, block_size>>>(bitstream_ptr, max_size);
}
__global__ void kernel_compute_gtli_tables(gpu_unpacked_info_t *infos,
                                           const uint8_t *sb_gains, const uint8_t *sb_priority,
                                           const uint32_t n_lvls, const uint32_t column_num, const uint32_t len)
{

    const uint32_t tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < len)
    {
        const uint32_t prec_y_idx = tid / (n_lvls * column_num);
        const uint32_t column = (tid / n_lvls) % column_num;
        const uint32_t lvl = tid % n_lvls;
        gpu_unpacked_info_t info_cur = infos[prec_y_idx * column_num + column];
        const int gain = sb_gains[lvl];
        const int scenario = info_cur.quantization;
        const int refinement = info_cur.refinement;
        int val = scenario - gain;
        const uint8_t add_1bp = (sb_priority[lvl] < refinement);
        if (add_1bp)
            val -= 1;
        val = MAX(val, 0);
        val = MIN(val, MAX_GCLI);
        info_cur.gtli_table_data[lvl] = info_cur.gtli_table_gcli[lvl] = val;
    }
}

inline int prec_y_idx_is_first_of_slice(const ids_t *ids, const uint32_t prec_y_idx, const uint32_t slice_height)
{
    assert(prec_y_idx >= 0 && slice_height > 0);
    return (((prec_y_idx * ids->ph) % slice_height) == 0);
}
inline int precinct_subpkt_of(const ids_t *ids, uint32_t position)
{
    return ids->pi[position].s;
}
__device__ __host__ void precinct_band_index_of(const ids_t *ids, uint32_t position, uint32_t *val)
{
    *val = ids->pi[position].b;
}
__device__ __host__ void precinct_ypos_of(const ids_t *ids, uint32_t position, uint32_t *val)
{
    *val = ids->pi[position].y - ids->l0[ids->pi[position].b];
}
__device__ __host__ void precinct_in_band_height_of(const ids_t *ids, const uint32_t prec_y_idx, uint32_t band_index, uint32_t *val)
{
    const int is_last_precinct_y = (prec_y_idx < (ids->npy - 1)) ? 0 : 1;
    *val = ids->l1[is_last_precinct_y][band_index] - ids->l0[band_index];
}
__device__ __host__ void precinct_is_line_present(const ids_t *ids, const uint32_t prec_y_idx, uint32_t lvl, uint32_t ypos, uint32_t *val)
{
    precinct_in_band_height_of(ids, prec_y_idx, lvl, val);
    *val = ypos < *val;
}
__device__ __host__ void precinct_gcli_width_of(uint32_t *gclis_sizes, int idx_from_level[MAX_PRECINCT_HEIGHT][MAX_PACKETS], uint32_t column, uint32_t npi, uint32_t band_index, uint32_t *val)
{
    int idx = idx_from_level[0][band_index];
    *val = gclis_sizes[column * npi + idx];
}
__device__ __host__ void precinct_gcli_offset_of(uint32_t *prefix_sum_of_size, int idx_from_level[MAX_PRECINCT_HEIGHT][MAX_PACKETS], uint32_t column, uint32_t npi, uint32_t band_index, uint32_t *val)
{
    int idx = idx_from_level[0][band_index];
    *val = prefix_sum_of_size[column * npi + idx];
}
__device__ void _device_read_n_bits(uint64_t *bitstream_ptr, uint64_t *bitstream_ptr_cur, uint32_t bit_offset,
                                    uint64_t max_size, uint64_t *val, uint16_t nbits)
{
    bitstream_ptr_cur += bit_offset / MAXB;
    bit_offset %= MAXB;
    uint64_t temp = 0;
    int available0 = MAXB - bit_offset;
    int len0 = (available0 >= nbits) ? nbits : available0;
    int len1 = nbits - len0;
    if (len0)
    {
        temp |= (((*bitstream_ptr_cur) >> (available0 - len0)) << len1);
    }
    if (len1)
    {
        uint64_t cur = 0UL;
        bitstream_ptr_cur++;
        if ((uint64_t)(bitstream_ptr_cur + 1 - bitstream_ptr) * 8 <= max_size)
        {
            cur = *bitstream_ptr_cur;
        }
        else
        {
            for (uint64_t i = 0; i < (max_size % 8); ++i)
            {
                ((uint8_t *)&cur)[i] = ((uint8_t *)(bitstream_ptr_cur))[i];
            }
        }
        bit_offset = 0;
        temp |= ((cur) >> (MAXB - len1));
    }
    if (nbits < 64)
    {
        *val = temp & ((1ULL << nbits) - 1ULL);
    }
}
__device__ void _device_gcli_method_is_enabled(uint32_t enabled, int gcli_method, int precinct_group, int *ret)
{
#define is_run_enabled(run) ((1ULL << (run)) & (enabled_runs))
    const uint32_t enabled_alphabets = (enabled >> METHOD_ENABLE_MASK_ALPHABETS_OFFSET) & ((1UL << ALPHABET_COUNT) - 1);
    const uint32_t enabled_predictions = (enabled >> METHOD_ENABLE_MASK_PREDICTIONS_OFFSET) & ((1UL << PRED_COUNT) - 1);
    const uint32_t enabled_runs = (enabled >> METHOD_ENABLE_MASK_RUNS_OFFSET) & ((1UL << RUN_COUNT) - 1);

    const int alphabet = method_get_alphabet(gcli_method);
    const int pred = method_get_pred(gcli_method);
    const int run = method_get_run(gcli_method);

    if (!((1ULL << alphabet) & enabled_alphabets))
    {
        *ret = 0;
        return;
    }

    if (alphabet != ALPHABET_RAW_4BITS)
    {
        if (!is_run_enabled(run))
        {
            *ret = 0;
            return;
        }

        if (!((1ULL << pred) & enabled_predictions))
        {
            *ret = 0;
            return;
        }

        if (precinct_group == PRECINCT_FIRST_OF_SLICE && pred != PRED_NONE)
        {
            *ret = 0;
            return;
        }
    }
    else
    {
        if (run != RUN_NONE || pred != PRED_NONE)
        {
            *ret = 0;
            return;
        }
    }
    *ret = 1;
    return;
}
__device__ void _device_gcli_method_get_signaling(int gcli_method, uint32_t enabled_methods, int *ret)
{
    int signaling = 0;
    if (gcli_method == method_get_idx(ALPHABET_RAW_4BITS, 0, 0))
    {
        *ret = -1;
        return;
    }
    const int uses_run = ((method_get_run(gcli_method) == RUN_SIGFLAGS_ZRF) || (method_get_run(gcli_method) == RUN_SIGFLAGS_ZRCSF));
    *ret = (uses_run ? 0x2 : 0) | ((method_get_pred(gcli_method) == PRED_VER) ? 0x1 : 0);
    return;
}
__global__ void kernel_read_gcli_sb_methods(gpu_unpacked_info_t *infos, uint64_t *bitstream_ptr,
                                            uint64_t max_size, const int enabled_methods,
                                            const uint32_t bands_count, const uint32_t column_num,
                                            const uint32_t len)
{
    const uint32_t tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < len)
    {
        const uint32_t prec_y_idx = tid / (bands_count * column_num);
        const uint32_t column = (tid / bands_count) % column_num;
        const uint32_t band_idx = tid % bands_count;
        uint64_t val;
        gpu_unpacked_info_t info = infos[prec_y_idx * column_num + column];
        uint64_t *bitstream_ptr_cur = bitstream_ptr + info.gcli_sb_methods_bitstream_info.ptr_diff;
        uint32_t bit_offset = info.gcli_sb_methods_bitstream_info.offset;
        _device_read_n_bits(bitstream_ptr, bitstream_ptr_cur, bit_offset + GCLI_METHOD_NBITS * band_idx, max_size, &val, GCLI_METHOD_NBITS);

        for (int gcli_method = 0; gcli_method < GCLI_METHODS_NB; gcli_method++)
        {
            int ret = 0;
            _device_gcli_method_is_enabled(enabled_methods, gcli_method, PRECINCT_ALL, &ret);
            if (ret)
            {
                _device_gcli_method_get_signaling(gcli_method, enabled_methods, &ret);
                if (ret == (int)val)
                {
                    info.gcli_sb_methods[band_idx] = gcli_method;
                    return;
                }
            }
        }
        info.gcli_sb_methods[band_idx] = -1;
        return;
    }
}
__global__ void kernel_read_inclusion_mask(gpu_unpacked_info_t *info_cur, uint64_t *bitstream_ptr, uint64_t *bitstream_ptr_cur,
                                           uint32_t bit_offset, uint32_t gcli_band_offset,
                                           uint64_t max_size,
                                           const uint32_t significance_group_size,
                                           const uint32_t len)
{
    uint32_t tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < len)
    {
        uint64_t val = 0;
        _device_read_n_bits(bitstream_ptr, bitstream_ptr_cur, bit_offset + tid, max_size, &val, 1);
        val = (!val) & 0x1;
        tid += gcli_band_offset;
        for (int i = 0; i < significance_group_size; i++)
        {
            info_cur->gpu_inclusion_mask[tid + i] = val;
        }
    }
}

__global__ void kernel_prepare_inclusion_mask(uint32_t *gclis_prefix_sum, uint32_t *gclis_sizes, gpu_unpacked_info_t *info_cur, int idx_from_level[MAX_PRECINCT_HEIGHT][MAX_PACKETS], uint64_t *bitstream_ptr,
                                              uint64_t max_size,
                                              ids_t *ids,
                                              const uint32_t significance_group_size,
                                              uint32_t prec_y_idx, uint32_t column, uint32_t subpkt,
                                              const uint32_t len)
{
    uint32_t tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < len)
    {
        const int idx = info_cur->line_idxs[subpkt].start_idx + tid;

        uint32_t lvl;
        precinct_band_index_of(ids, idx, &lvl);
        uint32_t ypos;
        precinct_ypos_of(ids, idx, &ypos);
        uint32_t is_present;
        precinct_is_line_present(ids, prec_y_idx, lvl, ypos, &is_present);
        if (is_present)
        {
            const int sb_gcli_method = info_cur->gcli_sb_methods[lvl];
            if (method_uses_sig_flags(sb_gcli_method))
            {
                const int block_size = BLOCK_SIZE;
                uint32_t gcli_width;
                // precinct_gcli_width_of(multi_buf_t * gclis_mb, int **idx_from_level, int band_index, int *val)
                precinct_gcli_width_of(gclis_sizes, idx_from_level, column, ids->npi, lvl, &gcli_width);
                uint64_t *bitstream_ptr_cur = info_cur->significance_bitstream_infos->ptr_diff + bitstream_ptr;
                uint32_t bit_offset = info_cur->significance_bitstream_infos->offset;
                uint32_t gcli_band_offset;
                precinct_gcli_offset_of(gclis_prefix_sum, idx_from_level, column, ids->npi, lvl, &gcli_band_offset);
                const int grid_size = (gcli_width / significance_group_size + block_size - 1) / block_size;
                kernel_read_inclusion_mask<<<grid_size, block_size>>>(info_cur, bitstream_ptr, bitstream_ptr_cur, bit_offset,
                                                                      gcli_band_offset, max_size, significance_group_size, gcli_width / significance_group_size);
            }
        }
    }
}

__global__ void kernel_unpack_gclis_significance(uint32_t *gclis_prefix_sum, uint32_t *gclis_sizes, gpu_unpacked_info_t *infos, int idx_from_level[MAX_PRECINCT_HEIGHT][MAX_PACKETS], uint64_t *bitstream_ptr,
                                                 ids_t *ids, const uint32_t significance_group_size, uint32_t nb_subpkts, uint32_t column_num, const uint32_t max_size, uint32_t len)
{
    const uint32_t tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < len)
    {

        const uint32_t subpkt = tid % nb_subpkts;
        const uint32_t column = (tid / nb_subpkts) % column_num;
        const uint32_t prec_y_idx = tid / (nb_subpkts * column_num);
        gpu_unpacked_info_t *info_cur = infos + prec_y_idx * column_num + column;
        const uint32_t block_size = BLOCK_SIZE;
        const uint32_t prepare_len = info_cur->line_idxs[subpkt].stop_idx - info_cur->line_idxs[subpkt].start_idx + 1;
        const uint32_t grid_size = (prepare_len + block_size - 1) / block_size; // ctx->xs_config->p.S_s
        kernel_prepare_inclusion_mask<<<grid_size, block_size>>>(gclis_prefix_sum, gclis_sizes, info_cur, idx_from_level, bitstream_ptr, max_size, ids, significance_group_size, prec_y_idx, column, subpkt, prepare_len);
    }
}

void gpu_unpack_gtlis_and_gclis_significance(gpu_xs_dec_context_t *ctx, gpu_unpacked_info_t *infos, const ids_t *ids,
                                             int nb_subpkts, const int n_precs, const int column_num)
{
    hipStream_t streams[2];
    hipStreamCreate(streams);
    hipStreamCreate(streams + 1);
    const uint32_t block_size = BLOCK_SIZE;
    const uint32_t bands_count = ids->nbands * n_precs * column_num;
    uint32_t grid_size = (bands_count + block_size - 1) / block_size;
    kernel_read_gcli_sb_methods<<<grid_size, block_size, 0, streams[0]>>>(infos, ctx->gpu_bitstream_ptr, ctx->max_size, ctx->enabled_methods, ids->nbands, column_num, bands_count);

    // put to different stream
    const uint32_t gtli_tables_len = ctx->level_count * n_precs * column_num;
    grid_size = (gtli_tables_len + block_size - 1) / block_size;
    kernel_compute_gtli_tables<<<grid_size, block_size, 0, streams[1]>>>(infos, ctx->gpu_lvl_gains, ctx->gpu_lvl_priorities, ctx->level_count, column_num, gtli_tables_len);

    hipDeviceSynchronize();

    const uint32_t significance_len = ctx->level_count * n_precs * column_num;
    grid_size = (significance_len + block_size - 1) / block_size;
    kernel_unpack_gclis_significance<<<grid_size, block_size>>>(ctx->gpu_gclis_prefix_sum, ctx->gpu_gclis_sizes, infos, ctx->idx_from_level, ctx->gpu_bitstream_ptr, ctx->gpu_ids, ctx->xs_config->p.S_s, nb_subpkts, column_num, ctx->max_size, significance_len);
    hipDeviceSynchronize();
    hipStreamDestroy(streams[0]);
    hipStreamDestroy(streams[1]);
}

__global__ void kernel_unpack_raw_gclis(uint64_t *bitstream_ptr, uint64_t *bitstream_ptr_cur, uint32_t bit_offset, uint64_t max_size, gcli_data_t *gclis, uint32_t gcli_width)
{
    const uint32_t tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < gcli_width)
    {
        uint64_t val;
        _device_read_n_bits(bitstream_ptr, bitstream_ptr_cur, bit_offset + tid * 4, max_size, &val, 4);
        gclis[tid] = (gcli_data_t)val;
    }
}

__device__ void _device_read_unary_unsigned(uint64_t *bitstream_ptr, uint64_t *bitstream_ptr_cur, uint32_t bit_offset, uint64_t max_size, int8_t *ret, uint32_t *n_bits)
{
    uint64_t bit = 1;
    int val = -1;
    while (bit)
    {
        _device_read_n_bits(bitstream_ptr, bitstream_ptr_cur, bit_offset++, max_size, &bit, 1);
        ++val;
    }
    *ret = val;
    *n_bits += val + 1;
}

__device__ void _device_read_unary_signed(uint64_t *bitstream_ptr, uint64_t *bitstream_ptr_cur, uint32_t bit_offset, uint64_t max_size, int8_t *ret, unary_alphabet_t alphabet, uint32_t *n_bits)
{
    int val = -1;
    uint32_t bit_offset_flag = bit_offset;
    switch (alphabet)
    {
    case UNARY_ALPHABET_FULL:
    {
        uint64_t bit = 1;
        do
        {
            _device_read_n_bits(bitstream_ptr, bitstream_ptr_cur, bit_offset++, max_size, &bit, 1);
            val++;
        } while (bit && val < 17);

        if (val == 1)
            val = -1;
        else if (val == 2)
            val = 1;
        else if (val == 3)
            val = -2;
        else if (val == 4)
            val = 2;
        else if (val > 4)
        {
            val -= 2;

            _device_read_n_bits(bitstream_ptr, bitstream_ptr_cur, bit_offset++, max_size, &bit, 1);
            if (bit)
                val = -val;
        }
        *ret = val;
        break;
    }
    case UNARY_ALPHABET_4_CLIPPED:
    {
        uint64_t bit = 1;
        do
        {
            _device_read_n_bits(bitstream_ptr, bitstream_ptr_cur, bit_offset++, max_size, &bit, 1);
            val++;
        } while (bit && val < 15);

        if (val == 1)
            val = -1;
        else if (val == 2)
            val = 1;
        else if (val == 3)
            val = -2;
        else if (val == 4)
            val = 2;

        if (val > 4)
        {
            val -= 2;
            if ((val) && (val != MAX_UNARY - 2))
                _device_read_n_bits(bitstream_ptr, bitstream_ptr_cur, bit_offset++, max_size, &bit, 1);
            if (bit)
                val = -val;
        }
        *ret = val;
        break;
    }
    case UNARY_ALPHABET_0:
    {
        uint64_t bit = 1;
        while (bit && (val < MAX_UNARY))
        {
            _device_read_n_bits(bitstream_ptr, bitstream_ptr_cur, bit_offset++, max_size, &bit, 1);
            val++;
        }
        if (val && (val != MAX_UNARY))
            _device_read_n_bits(bitstream_ptr, bitstream_ptr_cur, bit_offset++, max_size, &bit, 1);
        if (bit)
            val = -val;
        *ret = val;
        break;
    }
    default:
        assert(!"invalid alphabet specified");
        return;
    }
    *n_bits += bit_offset - bit_offset_flag;
}
__device__ void _device_read_bounded_code(uint64_t *bitstream_ptr, uint64_t *bitstream_ptr_cur, uint32_t bit_offset, uint64_t max_size,
                                          int8_t min_allowed, int8_t max_allowed,
                                          int8_t *val, uint32_t *n_bits)
{
    int8_t tmp;

    const int trigger = abs(min_allowed);

    _device_read_unary_unsigned(bitstream_ptr, bitstream_ptr_cur, bit_offset, max_size, &tmp, n_bits);
    if (tmp > 2 * trigger)
    {
        *val = tmp - trigger;
    }
    else
    {
        *val = (tmp + 1) / 2;
        if (tmp % 2)
        {
            *val = -*val;
        }
    }
}
__device__ void _device_unary_decode2(uint64_t *bitstream_ptr, uint64_t *bitstream_ptr_cur, uint32_t bit_offset, uint64_t max_size, gcli_pred_t *gcli_pred_buf, uint32_t *inclusion_mask,
                                      int len, int no_sign, int sb_gcli_method, unary_alphabet_t alph, uint32_t *n_bits)
{
    for (int i = 0; i < len; i++)
    {
        if (!method_uses_sig_flags(sb_gcli_method) || inclusion_mask[i])
        {
            if (!no_sign)
            {
                _device_read_unary_signed(bitstream_ptr, bitstream_ptr_cur, bit_offset + *n_bits, max_size, gcli_pred_buf + i, alph, n_bits);
            }
            else
            {
                _device_read_unary_unsigned(bitstream_ptr, bitstream_ptr_cur, bit_offset + *n_bits, max_size, gcli_pred_buf + i, n_bits);
            }
        }
        else
        {
            gcli_pred_buf[i] = 0;
        }
    }
}

__device__ void _device_bounded_decode2(uint64_t *bitstream_ptr, uint64_t *bitstream_ptr_cur, uint32_t bit_offset, uint64_t max_size,
                                        gcli_pred_t *gcli_pred_buf, uint32_t *inclusion_mask, gcli_data_t *gcli_top_buf,
                                        int sb_gcli_method, int gtli, int gtli_top, int len,
                                        uint32_t *n_bits)
{

    for (int i = 0; i < len; i++)
    {
        if (!method_uses_sig_flags(sb_gcli_method) || inclusion_mask[i])
        {
            int min_value = -20;
            int max_value = 20;
            if (gcli_top_buf)
            {
                int predictor = MAX(gcli_top_buf[i], MAX(gtli, gtli_top));
                min_value = -MAX(predictor - gtli, 0);
                max_value = MAX(MAX_GCLI - MAX(predictor, gtli), 0);
            }

            _device_read_bounded_code(bitstream_ptr, bitstream_ptr_cur, bit_offset, max_size, min_value, max_value, &gcli_pred_buf[i], n_bits);
        }
        else
        {
            gcli_pred_buf[i] = 0;
        }
    }
}
__global__ void kernel_tco_pred_ver_inverse(gcli_data_t *gclis_top, gcli_data_t *gclis, uint32_t *inclusion_mask, const int gtli, const int gtli_top, const int sig_flags_are_zrcsf, uint32_t len)
{
    const uint32_t tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < len)
    {
        int top = MAX(gclis_top[tid], MAX(gtli, gtli_top));
        gclis[tid] = top + gclis[tid];

        if (gclis[tid] <= gtli)
        {
            gclis[tid] -= gtli;
        }
        if (sig_flags_are_zrcsf && !inclusion_mask[tid])
        {
            gclis[tid] = 0;
        }
    }
}

__global__ void kernel_tco_pred_none_inverse(gcli_data_t *gclis, const int gtli, uint32_t len)
{
    const uint32_t tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < len)
    {
        gclis[tid] = (gclis[tid] > 0) ? gclis[tid] + gtli : 0;
    }
}
__global__ void kernel_unpack_gclis(gpu_unpacked_info_t *infos, uint64_t *bitstream_ptr, uint64_t max_size, uint32_t *gclis_prefix_sum, uint32_t *gclis_sizes,
                                    int idx_from_level[MAX_PRECINCT_HEIGHT][MAX_PACKETS],
                                    uint32_t len,
                                    const uint32_t slice_height, const uint32_t column_num,
                                    uint32_t nb_subpkts, const ids_t *gpu_ids)
{
    const uint32_t tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < len)
    {
        const uint32_t column = tid % column_num;
        const uint32_t slice_prec_y_idx = slice_height * (tid / column_num); //????
        const uint32_t bands_count = gpu_ids->nbands;
        int *gtli_table_top = NULL;
        gcli_data_t *gclis_top = NULL;
        for (int line_idx = 0; line_idx < slice_height; line_idx += gpu_ids->ph)
        {
            const int is_first_of_slice = !line_idx;
            const int prec_y_idx = ((line_idx + slice_prec_y_idx) / gpu_ids->ph);

            gpu_unpacked_info_t unpack_cur = infos[tid];
            for (int subpkt = 0; subpkt < nb_subpkts; subpkt++)
            {
                const int uses_raw_fallback = unpack_cur.uses_raw_fallback[subpkt];
                const int idx_start = unpack_cur.line_idxs[subpkt].start_idx;
                const int idx_stop = unpack_cur.line_idxs[subpkt].stop_idx;
                uint64_t *bitstream_ptr_cur = bitstream_ptr + unpack_cur.gcli_bitstream_infos[subpkt].ptr_diff;
                uint32_t bit_offset = unpack_cur.gcli_bitstream_infos[subpkt].offset;
                for (int idx = idx_start; idx <= idx_stop; idx++)
                {
                    uint32_t lvl;
                    precinct_band_index_of(gpu_ids, idx, &lvl);
                    uint32_t ypos;
                    precinct_ypos_of(gpu_ids, idx, &ypos);
                    if (ypos == 0 && !is_first_of_slice)
                    {

                        const int is_last_precinct_y = ((prec_y_idx - 1) < (gpu_ids->npy - 1)) ? 0 : 1;
                        const int ylast = gpu_ids->l1[is_last_precinct_y][lvl] - gpu_ids->l0[lvl] - 1;
                        int temp_idx = idx_from_level[ylast][lvl];
                        uint32_t gcli_offset_top;
                        precinct_gcli_offset_of(gclis_prefix_sum, idx_from_level, column, gpu_ids->npi, temp_idx, &gcli_offset_top);
                        gclis_top += gcli_offset_top;
                    }
                    else if (ypos != 0)
                    {
                        const int ylast = ypos - 1;
                        int temp_idx = idx_from_level[ylast][lvl];
                        uint32_t gcli_offset_top;
                        precinct_gcli_offset_of(gclis_prefix_sum, idx_from_level, column, gpu_ids->npi, temp_idx, &gcli_offset_top);
                        gclis_top = unpack_cur.gpu_gcli_buf + gcli_offset_top;
                    }
                    else
                    {
                        gclis_top = NULL;
                    }
                    uint32_t is_present;
                    precinct_is_line_present(gpu_ids, prec_y_idx, lvl, ypos, &is_present);
                    if (is_present)
                    {
                        int sb_gcli_method = uses_raw_fallback ? method_get_idx(ALPHABET_RAW_4BITS, 0, 0) : unpack_cur.gcli_sb_methods[gpu_ids->pi[idx].b];
                        const int gtli = unpack_cur.gtli_table_gcli[lvl];
                        const int gtli_top = (ypos == 0) ? ((gtli_table_top != NULL) ? gtli_table_top[lvl] : gtli) : (gtli);
                        uint32_t gcli_width;
                        precinct_gcli_width_of(gclis_sizes, idx_from_level, column, gpu_ids->npi, lvl, &gcli_width);
                        uint32_t gcli_offset;
                        precinct_gcli_offset_of(gclis_prefix_sum, idx_from_level, column, gpu_ids->npi, lvl, &gcli_offset);
                        gcli_data_t *gclis = unpack_cur.gpu_gcli_buf + gcli_offset;
                        uint32_t *inclusion_mask = unpack_cur.gpu_inclusion_mask + gcli_offset;

                        unary_alphabet_t alph = FIRST_ALPHABET;

                        if (method_is_raw(sb_gcli_method))
                        {
                            const int block_size = BLOCK_SIZE;
                            const int grid_size = (gcli_width + block_size - 1) / block_size;
                            kernel_unpack_raw_gclis<<<grid_size, block_size>>>(bitstream_ptr, bitstream_ptr_cur, bit_offset, max_size, gclis, gcli_width);
                            bit_offset += gcli_width << 2;
                        }
                        else
                        {
                            uint32_t n_bits = 0;
                            int no_prediction = method_uses_no_pred(sb_gcli_method) || ((method_uses_ver_pred(sb_gcli_method) && method_get_alphabet(sb_gcli_method) == ALPHABET_UNARY_UNSIGNED_BOUNDED) && (gclis_top == NULL));
                            int sig_flags_are_zrcsf = (method_get_run(sb_gcli_method) == RUN_SIGFLAGS_ZRCSF);
                            if ((method_get_alphabet(sb_gcli_method) != ALPHABET_UNARY_UNSIGNED_BOUNDED) || no_prediction)
                            {
                                _device_unary_decode2(bitstream_ptr, bitstream_ptr_cur, bit_offset, max_size, gclis, inclusion_mask, gcli_width, no_prediction, sb_gcli_method, alph, &n_bits);
                            }
                            else
                            {
                                _device_bounded_decode2(bitstream_ptr, bitstream_ptr_cur, bit_offset, max_size, gclis, inclusion_mask, gclis_top, sb_gcli_method, gtli, gtli_top, gcli_width, &n_bits);
                            }
                            bit_offset += n_bits;
                            if ((method_uses_ver_pred(sb_gcli_method) && gclis_top))
                            {
                                const int block_size = BLOCK_SIZE;
                                const int grid_size = (gcli_width + block_size - 1) / block_size;
                                kernel_tco_pred_ver_inverse<<<grid_size, block_size>>>(gclis_top, gclis, inclusion_mask, gtli, gtli_top, sig_flags_are_zrcsf, gcli_width);
                            }
                            else if (method_uses_no_pred(sb_gcli_method) || (method_uses_ver_pred(sb_gcli_method) && gclis_top == NULL))
                            {
                                const int block_size = BLOCK_SIZE;
                                const int grid_size = (gcli_width + block_size - 1) / block_size;
                                kernel_tco_pred_none_inverse<<<grid_size, block_size>>>(gclis, gtli, gcli_width);
                            }
                        }
                    }
                }
            }

            gtli_table_top = unpack_cur.gtli_table_gcli;
            gclis_top = unpack_cur.gpu_gcli_buf;
        }
    }
}

void gpu_unpack_gclis(gpu_xs_dec_context_t *ctx, gpu_unpacked_info_t *infos, const uint32_t column_num)
{
    const uint32_t block_size = BLOCK_SIZE;
    const uint32_t slice_height = ctx->xs_config->p.slice_height;
    const uint32_t slice_size = SIGFLAGS_NEXTLVL_SIZE(ctx->ids.h, slice_height) * column_num;
    const uint32_t grid_size = (slice_size + block_size - 1) / block_size;
    kernel_unpack_gclis<<<grid_size, block_size>>>(infos, ctx->gpu_bitstream_ptr, ctx->max_size, ctx->gpu_gclis_prefix_sum, ctx->gpu_gclis_sizes, ctx->idx_from_level, slice_size, slice_height, column_num, ctx->nb_subpkts, ctx->gpu_ids);
}
__global__ void kernel_unpack_data(uint64_t *bitstream_ptr, uint64_t *bitstream_ptr_cur, uint32_t bit_offset, uint64_t max_size,
                                   gcli_data_t *gclis, uint32_t *inclusion_mask,
                                   const int group_size, const int gtli, const uint8_t sign_packing,
                                   xs_data_in_t *image, uint64_t dst_inc, const int quant_type, const uint32_t len)
{
    const uint32_t tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < len)
    {
        // tid is
        const int i = tid % group_size;
        const int group = tid / group_size;
        const int gcli = gclis[group];
        uint32_t ret = 0;
        if (gcli > gtli)
        {
            uint64_t val;

            bit_offset += inclusion_mask[group - 1] * group_size;

            for (int bp = 0; bp < gcli - gtli; bp++)
            {
                _device_read_n_bits(bitstream_ptr, bitstream_ptr_cur, bit_offset + group_size + i + group_size * bp, max_size, &val, 1);
                ret |= (sig_mag_data_t)((val & 0x01) << (gcli - 1 - bp));
            }
            if (sign_packing == 0)
            {
                _device_read_n_bits(bitstream_ptr, bitstream_ptr_cur, bit_offset + i, max_size, &val, 1);
                ret |= (sig_mag_data_t)val << SIGN_BIT_POSITION;

                // dequant
                if (quant_type == 1)
                {
                    int sign = (ret & SIGN_BIT_MASK);
                    int phi = ret & ~SIGN_BIT_MASK;
                    int zeta = gcli - gtli + 1;
                    int rho = 0;
                    for (rho = 0; phi > 0; phi >>= zeta)
                        rho += phi;
                    ret = sign | rho;
                }
                else if (quant_type == 0)
                {
                    if (gtli > 0 && (ret & ~SIGN_BIT_MASK))
                        ret |= (1 << (gtli - 1));
                }
            }
        }
        image[tid * dst_inc] = ret;
    }
}

__global__ void kernel_unpack_sign(uint64_t *bitstream_ptr, uint64_t *bitstream_ptr_cur, uint32_t bit_offset, uint64_t max_size,
                                   gcli_data_t *gclis, uint32_t *inclusion_mask, const int group_size, const int gtli, const uint8_t sign_packing,
                                   xs_data_in_t *image, uint64_t dst_inc, const int quant_type, const uint32_t len)
{
    const uint32_t tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < len)
    {
        // tid is
        const int i = tid % group_size;
        const int group = tid / group_size;
        const int gcli = gclis[group];
        uint32_t ret = image[tid * dst_inc];
        if (ret)
        {
            uint64_t val;

            bit_offset += inclusion_mask[group - 1] * group_size;

            _device_read_n_bits(bitstream_ptr, bitstream_ptr_cur, bit_offset + i, max_size, &val, 1);
            ret |= (sig_mag_data_t)((val & 0x01) << SIGN_BIT_POSITION);

            // dequant
            if (quant_type == 1)
            {
                int sign = (ret & SIGN_BIT_MASK);
                int phi = ret & ~SIGN_BIT_MASK;
                int zeta = gcli - gtli + 1;
                int rho = 0;
                for (rho = 0; phi > 0; phi >>= zeta)
                    rho += phi;
                ret = sign | rho;
            }
            else if (quant_type == 0)
            {
                if (gtli > 0 && (ret & ~SIGN_BIT_MASK))
                    ret |= (1 << (gtli - 1));
            }
            image[tid * dst_inc] = ret;
        }
    }
}
// __global__ void kernel_prepare_data(bit_unpacker_t *bitstream, sig_mag_data_t *buf, int buf_len, gcli_data_t *gclis, int group_size, int gtli, const uint8_t sign_packing)
// {
//     const uint32_t tid = threadIdx.x + blockIdx.x * blockDim.x;
//     if (tid < len)
//     {
//     }
// }

__device__ __host__ void precinct_ptr_for_line_of_band(const ids_t *ids, xs_image_t *image,
                                                       const int band_idx, const int in_band_ypos,
                                                       const uint32_t prec_y_idx, const uint32_t column, const uint32_t is_last_column,
                                                       xs_data_in_t **ptr, uint64_t *x_inc, uint64_t *len)
{

    const int c = ids->band_idx_to_c_and_b[band_idx].c;
    const int b = ids->band_idx_to_c_and_b[band_idx].b;

    // Handle precinct component base.
    xs_data_in_t *the_ptr = image->comps_array[c];
    // Handle start y position of band.
    the_ptr += ids->band_is_high[b].y * ((uint64_t)1 << (ids->band_d[c][b].y - 1)) * ids->comp_w[c];
    // Handle start x position of band.
    the_ptr += ids->band_is_high[b].x * ((uint64_t)1 << (ids->band_d[c][b].x - 1));

    // Handle precinct y index.
    the_ptr += (uint64_t)ids->comp_w[c] * (ids->ph >> (image->sy[c] - 1)) * prec_y_idx;
    // Go to ypos line in precinct.
    the_ptr += (uint64_t)ids->comp_w[c] * in_band_ypos * ((uint64_t)1 << ids->band_d[c][b].y);

    // Handle precinct x index.
    the_ptr += (ids->pw[0] >> (image->sx[c] - 1)) * column;

    *ptr = the_ptr;                            // first sample
    *x_inc = 1ull << ids->band_d[c][b].x;      // increment to next sample
    *len = ids->pwb[is_last_column][band_idx]; // number of samples in line of band in precinct
}

__global__ void kernel_unpack_data_and_sign(uint64_t *bitstream_ptr, const uint64_t max_size, gpu_unpacked_info_t *infos,
                                            uint32_t *gclis_prefix_sum, uint32_t *gclis_sizes, int idx_from_level[MAX_PRECINCT_HEIGHT][MAX_PACKETS],
                                            const uint32_t nb_subpkts, const uint32_t column_num, const int sign_packing,
                                            const uint32_t group_size, const uint32_t len, const int quant_type,
                                            xs_image_t *gpu_image, ids_t *gpu_ids)
{
    const uint32_t tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < len)
    {
        const uint32_t subpkt = tid % nb_subpkts;
        const uint32_t column = (tid / nb_subpkts) % column_num;
        const uint32_t prec_y_idx = tid / (nb_subpkts * column_num);
        gpu_unpacked_info_t *info_cur = infos + prec_y_idx * column_num + column;
        uint32_t start_idx = info_cur->line_idxs[subpkt].start_idx;
        uint32_t stop_idx = info_cur->line_idxs[subpkt].stop_idx;
        uint32_t gcli_width;
        uint32_t gcli_offset;
        uint64_t *bitstream_ptr_cur = info_cur->data_bitstream_infos[subpkt].ptr_diff + bitstream_ptr;
        uint32_t bit_offset = info_cur->data_bitstream_infos[subpkt].offset;

        for (int idx = start_idx; idx <= stop_idx; idx++)
        {
            uint32_t lvl;
            precinct_band_index_of(gpu_ids, idx, &lvl);
            uint32_t ypos;
            precinct_ypos_of(gpu_ids, idx, &ypos);
            uint32_t is_present;
            precinct_is_line_present(gpu_ids, prec_y_idx, lvl, ypos, &is_present);
            if (is_present)
            {
                const int gtli = info_cur->gtli_table_gcli[lvl];

                precinct_gcli_width_of(gclis_sizes, idx_from_level, column, gpu_ids->npi, lvl, &gcli_width);
                precinct_gcli_offset_of(gclis_prefix_sum, idx_from_level, column, gpu_ids->npi, lvl, &gcli_offset);

                gcli_data_t *gclis = gcli_width + info_cur->gpu_gcli_buf;
                uint8_t offset_width = (sign_packing == 0) ? group_size : 0;
                uint32_t *inclusion_mask = info_cur->gpu_inclusion_mask + gcli_offset;
                inclusion_mask[0] = (gclis[0] > gtli) ? (gclis[0] - gtli + offset_width) : 0;

                for (int group = 1; group < gcli_width; group++)
                {
                    inclusion_mask[group] = inclusion_mask[group - 1];
                    if (gclis[group] > gtli)
                    {
                        inclusion_mask[group] += gclis[group] - gtli + offset_width;
                    }
                }
                uint32_t bits_sum = inclusion_mask[gcli_width - 1];
                const int coff_width = gcli_width * group_size;
                xs_data_in_t *dst;
                uint64_t dst_inc, dst_len;
                uint32_t is_last_column = (column == gpu_ids->npx - 1) ? 1 : 0;
                precinct_ptr_for_line_of_band(gpu_ids, gpu_image, lvl, ypos, prec_y_idx, column, is_last_column, &dst, &dst_inc, &dst_len);
                int block_size = BLOCK_SIZE;
                int grid_size = (coff_width + block_size - 1) / block_size;
                kernel_unpack_data<<<grid_size, block_size>>>(bitstream_ptr, bitstream_ptr_cur, bit_offset, max_size, gclis, inclusion_mask, group_size, gtli, sign_packing, dst, dst_inc, quant_type, coff_width);

                if (sign_packing)
                {
                    inclusion_mask = (uint32_t *)malloc(sizeof(uint32_t) * coff_width);
                    inclusion_mask[0] = (dst[0] != 0) ? 1 : 0;

                    for (int group = 1; group < coff_width; group++)
                    {
                        inclusion_mask[group] = inclusion_mask[group - 1];
                        if (dst[group * dst_inc] != 0)
                        {
                            inclusion_mask[group]++;
                        }
                    }
                    grid_size = (coff_width + block_size - 1) / block_size;
                    kernel_unpack_sign<<<grid_size, block_size>>>(bitstream_ptr, bitstream_ptr_cur, bit_offset, max_size, gclis, inclusion_mask, group_size, gtli, sign_packing, dst, dst_inc, quant_type, coff_width);
                    free(inclusion_mask);
                }
                bit_offset += bits_sum;
            }
        }
    }
}
void gpu_unpack_data_and_sign(gpu_xs_dec_context_t *ctx, gpu_unpacked_info_t *infos, xs_image_t *gpu_image, const uint32_t column_num)
{
    const uint32_t block_size = BLOCK_SIZE;
    const uint32_t n_precs = SIGFLAGS_NEXTLVL_SIZE(ctx->ids.h, ctx->ids.ph) * column_num;
    const uint32_t grid_size = (n_precs + block_size - 1) / block_size;
    kernel_unpack_data_and_sign<<<grid_size, block_size>>>(ctx->gpu_bitstream_ptr, ctx->max_size, infos,
                                                           ctx->gpu_gclis_prefix_sum, ctx->gpu_gclis_sizes, ctx->idx_from_level,
                                                           ctx->nb_subpkts, column_num, ctx->use_sign_subpkt,
                                                           ctx->group_size, n_precs, ctx->quant_type,
                                                           gpu_image, ctx->gpu_ids);
}
void unpack_bit_offset_info(gpu_xs_dec_context_t *ctx, bit_unpacker_t *bitstream, gpu_unpacked_info_t *info, gpu_unpacked_info_t *gpu_info,
                            uint32_t prec_y_idx, uint32_t column, int extra_bits_before_precinct)
{
    uint64_t val;
    int empty;
    int len_before_subpkt = 0;
    int gcli_sb_methods[MAX_NBANDS];
    int subpkt = 0;
    int subpkt_len;

    const uint32_t position_count = ctx->ids.npi;
    const bool use_long_precinct_headers = ctx->ids.use_long_precinct_headers;
    const int bitpos_prc_start = (int)bitunpacker_consumed_bits(bitstream);

    if (ctx->xs_config->verbose > 2)
    {
        fprintf(stderr, "(bitpos=%d) Precinct (bytepos=%d)\n", bitpos_prc_start, bitpos_prc_start >> 3);
    }
    hipDeviceSynchronize();
    if (!ctx->is_init)
    {
        hipMalloc((void **)&(info->gpu_inclusion_mask), sizeof(uint32_t) * 1);
        hipMalloc((void **)&(info->gpu_gcli_buf), sizeof(gcli_type_t) * 1);
    }
    // Start of precinct.
    bitunpacker_read(bitstream, &val, PREC_HDR_PREC_SIZE);
    const int Lprc = info->Lprc = ((int)val << 3);
    bitunpacker_read(bitstream, &val, PREC_HDR_QUANTIZATION_SIZE);
    const int quantization = info->quantization = (int)val;
    bitunpacker_read(bitstream, &val, PREC_HDR_REFINEMENT_SIZE);
    const int refinement = info->refinement = (int)val;

    bitunpacker_set_info(bitstream, &(info->gcli_sb_methods_bitstream_info));
    const int bands_count = ctx->level_count;
    bitunpacker_skip(bitstream, bands_count * GCLI_METHOD_NBITS);
    // for (int band = 0; band < bands_count; ++band)
    // {
    //     bitunpacker_read(bitstream, &val, GCLI_METHOD_NBITS);
    //     gcli_sb_methods[band] = gcli_method_from_signaling((int)val, ctx->enabled_methods);
    // }
    bitunpacker_align(bitstream, PREC_HDR_ALIGNMENT);
    const int bitpos_at_prc_data = (int)bitunpacker_consumed_bits(bitstream);
#ifdef PACKING_GENERATE_FRAGMENT_CODE
    // Add precinct header bits.
    extra_bits_before_precinct += bitpos_at_prc_data - bitpos_prc_start;
#endif

    if (ctx->xs_config->verbose > 3)
    {
        fprintf(stderr, "(bitpos=%d) precinct header read (prec_len=%d quant=(%d,%d)\n", bitpos_at_prc_data, Lprc, quantization, refinement);
    }
    // compute gtli
    // gpu_compute_gtli_tables(quantization, refinement, ctx->level_count, ctx->xs_config->p.lvl_gains, ctx->xs_config->p.lvl_priorities, ctx->gtli_table_data, ctx->gtli_table_gcli, &empty);

    for (int idx_start = 0, idx_stop = 0; idx_stop < position_count; idx_stop++)
    {
        if ((idx_stop != (position_count - 1)) && (precinct_subpkt_of(&ctx->ids, idx_stop) == precinct_subpkt_of(&ctx->ids, idx_stop + 1)))
        {
            continue;
        }
        uint32_t lvl;
        precinct_band_index_of(&ctx->ids, idx_start, &lvl);
        uint32_t ypos;
        precinct_ypos_of(&ctx->ids, idx_start, &ypos);
        uint32_t is_present;
        precinct_is_line_present(&ctx->ids, prec_y_idx, lvl, ypos, &is_present);
        if (!is_present)
        {
            ++subpkt;
            idx_start = idx_stop + 1;
            continue;
        }

#ifdef PACKING_GENERATE_FRAGMENT_CODE
        const int bitpos_packet_start = (int)bitunpacker_consumed_bits(bitstream);
#endif

        // Start of packet.
        bitunpacker_read(bitstream, &val, 1);
        info->uses_raw_fallback[subpkt] = (int)val & 0x1;

        bitunpacker_read(bitstream, &val, use_long_precinct_headers ? PKT_HDR_DATA_SIZE_LONG : PKT_HDR_DATA_SIZE_SHORT);
        info->data_len[subpkt] = (int)val;

        bitunpacker_read(bitstream, &val, use_long_precinct_headers ? PKT_HDR_GCLI_SIZE_LONG : PKT_HDR_GCLI_SIZE_SHORT);
        info->gcli_len[subpkt] = (int)val;

        bitunpacker_read(bitstream, &val, use_long_precinct_headers ? PKT_HDR_SIGN_SIZE_LONG : PKT_HDR_SIGN_SIZE_SHORT);
        info->sign_len[subpkt] = (int)val;

        bitunpacker_align(bitstream, PKT_HDR_ALIGNMENT);

        if (ctx->xs_config->verbose > 2)
        {
            const int bitpos = (int)bitunpacker_consumed_bits(bitstream);
            fprintf(stderr, "(bitpos=%d) Subpacket DATALEN=%d GCLILEN=%d SIGNLEN=%d (force_raw%d)\n", bitpos,
                    info->data_len[subpkt],
                    info->gcli_len[subpkt],
                    info->sign_len[subpkt],
                    info->uses_raw_fallback[subpkt]);
        }

        // get significance group bitstreamm info
        bitunpacker_set_info(bitstream, info->significance_bitstream_infos + subpkt);
        if (!info->uses_raw_fallback[subpkt])
        {
            int skip_bits = 0;
            for (int idx = idx_start; idx <= idx_stop; idx++)
            {
                uint32_t lvl;
                precinct_band_index_of(&ctx->ids, idx, &lvl);
                uint32_t ypos;
                precinct_ypos_of(&ctx->ids, idx, &ypos);
                uint32_t is_present;
                precinct_is_line_present(&ctx->ids, prec_y_idx, lvl, ypos, &is_present);
                if (is_present)
                {
                    uint32_t gcli_width;
                    precinct_gcli_width_of(ctx->gclis_sizes, ctx->idx_from_level, column, ctx->ids.npi, lvl, &gcli_width);
                    const int significance_group_size = ctx->xs_config->p.S_s;
                    skip_bits += SIGFLAGS_NEXTLVL_SIZE(gcli_width, significance_group_size);
                }
            }
            bitunpacker_skip(bitstream, (skip_bits));
        }
        bitunpacker_align(bitstream, SUBPKT_ALIGNMENT);

        // get gcli bitstreamm info
        len_before_subpkt = (int)bitunpacker_consumed_bits(bitstream);
        bitunpacker_set_info(bitstream, info->gcli_bitstream_infos + subpkt);
        bitunpacker_skip(bitstream, (info->gcli_len[subpkt] << 3));
        bitunpacker_align(bitstream, SUBPKT_ALIGNMENT);

        // get data bitstreamm info
        bitunpacker_set_info(bitstream, info->data_bitstream_infos + subpkt);
        bitunpacker_skip(bitstream, (info->data_len[subpkt] << 3));
        bitunpacker_align(bitstream, SUBPKT_ALIGNMENT);

        // get sign bitstreamm info
        bitunpacker_set_info(bitstream, info->sign_bitstream_infos + subpkt);
        bitunpacker_skip(bitstream, (info->sign_len[subpkt] << 3));
        bitunpacker_align(bitstream, SUBPKT_ALIGNMENT);

#ifdef PACKING_GENERATE_FRAGMENT_CODE
        if (ctx->fragment_info_cb != NULL)
        {
            int n_gclis = 0;
            for (int idx = idx_start; idx <= idx_stop; idx++)
            {
                uint32_t val;
                precinct_band_index_of(&ctx->ids, idx, &val);
                precinct_gcli_width_of(ctx->gclis_sizes, ctx->idx_from_level, column, ctx->ids.npi, lvl, &val);
                n_gclis += val;
            }

            const int bitpos_packet_end = (int)bitunpacker_consumed_bits(bitstream);
            // account for EOC if really last fragment (the decoder will verify the actual EOC being present)
            const int _fragment_size = bitpos_packet_end - bitpos_packet_start + extra_bits_before_precinct;
            extra_bits_before_precinct = 0;
            ctx->fragment_info_cb(ctx->fragment_info_context, ctx->fragment_cnt, _fragment_size, n_gclis, 0);
            ++ctx->fragment_cnt;
        }
#endif

        ++subpkt;
        idx_start = idx_stop + 1;
    }
    ctx->nb_subpkts = subpkt;
    hipMemcpyAsync(gpu_info, info, sizeof(gpu_unpacked_info_t), hipMemcpyHostToDevice);
    const int padding_len = Lprc - ((int)bitunpacker_consumed_bits(bitstream) - bitpos_at_prc_data);
    assert(padding_len >= 0);
#ifdef PACKING_GENERATE_FRAGMENT_CODE
    if (ctx->fragment_info_cb != NULL && padding_len > 0)
    {
        // Late contribution (the padding), so refer to last fragment ID (value -1).
        ctx->fragment_info_cb(ctx->fragment_info_context, -1, 0, 0, padding_len);
    }
#endif
    bitunpacker_skip(bitstream, padding_len);
}

gpu_xs_dec_context_t *gpu_xs_dec_init(xs_config_t *xs_config, xs_image_t *image)
{
    gpu_xs_dec_context_t *ctx;

    ctx = (gpu_xs_dec_context_t *)malloc(sizeof(gpu_xs_dec_context_t));
    if (!ctx)
    {
        return NULL;
    }

    memset(ctx, 0, sizeof(gpu_xs_dec_context_t));
    ctx->xs_config = xs_config;

    assert(xs_config != NULL);
    if (!xs_config_validate(xs_config, image))
    {
        free(ctx);
        return NULL;
    }
    ctx->quant_type = xs_config->p.Qpih;
    ctx->group_size = xs_config->p.N_g;
    ids_construct(&ctx->ids, image, xs_config->p.NLx, xs_config->p.NLy, xs_config->p.Sd, xs_config->p.Cw, xs_config->p.Lh);
    hipMalloc((void **)&ctx->gpu_ids, sizeof(ids_t));
    hipMemcpyAsync(ctx->gpu_ids, &ctx->ids, sizeof(ids_t), hipMemcpyHostToDevice);
    ctx->level_count = ctx->ids.nbands;
    ctx->gclis_sizes = (uint32_t *)malloc(sizeof(uint32_t) * (ctx->ids.npx * ctx->ids.npi));
    ctx->gclis_prefix_sum = (uint32_t *)malloc(sizeof(uint32_t) * (ctx->ids.npx * ctx->ids.npi));
    hipMalloc((void **)&ctx->gpu_gclis_sizes, sizeof(uint32_t) * (ctx->ids.npx * ctx->ids.npi));
    hipMalloc((void **)&ctx->gpu_gclis_prefix_sum, sizeof(uint32_t) * (ctx->ids.npx * ctx->ids.npi));
    for (int column = 0; column < ctx->ids.npx; column++)
    {

        int is_last_column = (column == ctx->ids.npx - 1) ? 1 : 0;
        for (int idx = 0; idx < ctx->ids.npi; ++idx)
        {
            const int band = ctx->ids.pi[idx].b;
            const int y = ctx->ids.pi[idx].y - ctx->ids.l0[band]; // relative Y in band

            ctx->idx_from_level[y][band] = idx;

            const int N_cg = (ctx->ids.pwb[is_last_column][band] + ctx->group_size - 1) / ctx->group_size;
            ctx->gclis_sizes[column * ctx->ids.npi + idx] = N_cg;
            ctx->gclis_prefix_sum[column * ctx->ids.npi + idx] = idx ? (ctx->gclis_sizes[column * ctx->ids.npi + idx - 1] + ctx->gclis_prefix_sum[column * ctx->ids.npi + idx - 1]) : 0;
        }
    }
    hipMemcpyAsync(ctx->gpu_gclis_sizes, ctx->gclis_sizes, sizeof(uint32_t) * (ctx->ids.npx * ctx->ids.npi), hipMemcpyHostToDevice);
    hipMemcpyAsync(ctx->gpu_gclis_prefix_sum, ctx->gclis_prefix_sum, sizeof(uint32_t) * (ctx->ids.npx * ctx->ids.npi), hipMemcpyHostToDevice);
    ctx->bitstream = bitunpacker_init();
    return ctx;
}

void gpu_unpack_bitstream_infos(gpu_xs_dec_context_t *ctx, void *bitstream_buf, uint64_t bitstream_buf_size, xs_image_t *image_out)
{
    int slice_idx = 0;
    uint64_t bitstream_pos = 0;
    static gpu_unpacked_info_t *gpu_infos = NULL;
    static gpu_unpacked_info_t *cpu_info = NULL;
    if (!ctx->is_init)
    {
        if (gpu_infos == NULL)
        {
            hipMalloc((void **)&gpu_infos, sizeof(gpu_unpacked_info_t) * ((ctx->ids.npx) * ((ctx->ids.h + ctx->ids.ph - 1) / ctx->ids.ph) + 10));
        }
        if (cpu_info == NULL)
        {
            cpu_info = (gpu_unpacked_info_t *)malloc(sizeof(gpu_unpacked_info_t));
        }
        if (ctx->gpu_bitstream_ptr == NULL)
        {
            hipMalloc((void **)&ctx->gpu_bitstream_ptr, (bitstream_buf_size + 8) & (~0x7));
            ctx->max_size = bitstream_buf_size;
        }
    }
    hipMemcpy(ctx->gpu_bitstream_ptr, bitstream_buf, (bitstream_buf_size + 8) & (~0x7), hipMemcpyHostToDevice);
    // memset(cpu_info, 0, sizeof(gpu_unpacked_info_t));
    gpu_convert_ipx_htobe64(ctx->gpu_bitstream_ptr, (((bitstream_buf_size + 8) & (~0x7)) / sizeof(uint64_t)));
    bitunpacker_set_buffer(ctx->bitstream, bitstream_buf, bitstream_buf_size);

    xs_parse_head(ctx->bitstream, NULL, NULL);

#ifdef PACKING_GENERATE_FRAGMENT_CODE
    memset(&ctx->fragment_info_buf, 0, sizeof(xs_buffering_fragment_t));
    ctx->fragment_info_buf.id = -1;
#endif
    ids_t *ids = &ctx->ids;

    uint32_t prec_y_idx;
    for (int line_idx = 0; line_idx < ids->h; line_idx += ids->ph)
    {
        prec_y_idx = (line_idx / ids->ph);
        for (int column = 0; column < ids->npx; column++)
        {
            const int is_first_of_slice = prec_y_idx_is_first_of_slice(ids, prec_y_idx, ctx->xs_config->p.slice_height);
            if (is_first_of_slice && column == 0)
            {
                int slice_idx_check;
                xs_parse_slice_header(ctx->bitstream, &slice_idx_check);
                assert(slice_idx_check == (slice_idx++));
                if (ctx->xs_config->verbose > 1)
                {
                    fprintf(stderr, "Read Slice Header (slice_idx=%d)\n", slice_idx_check);
                }
            }
#ifdef PACKING_GENERATE_FRAGMENT_CODE
            const int extra_bits_before_precinct = (int)(bitunpacker_consumed_bits(ctx->bitstream) - bitstream_pos);
#else
            const int extra_bits_before_precinct = 0;
#endif
            unpack_bit_offset_info(ctx, ctx->bitstream, cpu_info, gpu_infos, prec_y_idx, column, extra_bits_before_precinct);
        }
    }
    const int n_precs = prec_y_idx;
    gpu_unpack_gtlis_and_gclis_significance(ctx, gpu_infos, &ctx->ids, ctx->nb_subpkts, n_precs, ids->npx);

    gpu_unpack_gclis(ctx, gpu_infos, ctx->ids.npx);

    ctx->is_init = 1;
}
